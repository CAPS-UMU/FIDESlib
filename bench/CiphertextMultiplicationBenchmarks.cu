//
// Created by oscar on 21/10/24.
//

#include <benchmark/benchmark.h>

#include "Benchmark.cuh"
#include "CKKS/KeySwitchingKey.cuh"

namespace FIDESlib::Benchmarks {
BENCHMARK_DEFINE_F(GeneralFixture, CiphertextMultiplication)(benchmark::State& state) {
    if (this->generalTestParams.multDepth <= static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L <= level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};
    std::vector<double> x2 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));
    lbcrypto::Plaintext ptxt2 = cc->MakeCKKSPackedPlaintext(x2, 1, state.range(3));

    ptxt1->SetLevel(state.range(3));
    ptxt2->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);
    auto c2 = cc->Encrypt(keys.publicKey, ptxt2);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    FIDESlib::CKKS::RawCipherText raw2 = FIDESlib::CKKS::GetRawCipherText(cc, c2);

    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);
    FIDESlib::CKKS::Ciphertext GPUct2(GPUcc, raw2);

    FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
    kskEval.Initialize(GPUcc, rawKskEval);

    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);
    for (auto _ : state) {
        GPUct1.mult(GPUct2, kskEval, false);
        CudaCheckErrorMod;
    }
    CudaCheckErrorMod;
}

BENCHMARK_DEFINE_F(GeneralFixture, CiphertextSquaring)(benchmark::State& state) {
    if (this->generalTestParams.multDepth <= static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L <= level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));

    ptxt1->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);

    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
    kskEval.Initialize(GPUcc, rawKskEval);

    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);
    for (auto _ : state) {
        GPUct1.square(kskEval, false);
        CudaCheckErrorMod;
    }
    CudaCheckErrorMod;
}

BENCHMARK_DEFINE_F(GeneralFixture, MultScalar)(benchmark::State& state) {
    if (this->generalTestParams.multDepth <= static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L <= level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));

    ptxt1->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);
    for (auto _ : state) {
        GPUct1.multScalar(1.01231331, false);
        CudaCheckErrorMod;
    }
    CudaCheckErrorMod;
}

BENCHMARK_REGISTER_F(GeneralFixture, CiphertextMultiplication)
    ->ArgsProduct({{0, 1, 2, 3, 6}, {0}, BATCH_CONFIG, LEVEL_CONFIG});
BENCHMARK_REGISTER_F(GeneralFixture, CiphertextSquaring)->ArgsProduct({{0, 1, 2, 3}, {0}, {2, 6, 12}, LEVEL_CONFIG});
BENCHMARK_REGISTER_F(GeneralFixture, MultScalar)->ArgsProduct({{0, 1, 2, 3}, {0}, BATCH_CONFIG, LEVEL_CONFIG});
}  // namespace FIDESlib::Benchmarks