//
// Created by carlosad on 19/11/24.
//
#include "Benchmark.cuh"
#include "CKKS/ApproxModEval.cuh"
#include "CKKS/Bootstrap.cuh"
#include "CKKS/BootstrapPrecomputation.cuh"
#include "CKKS/CoeffsToSlots.cuh"
#include "CKKS/KeySwitchingKey.cuh"

namespace FIDESlib::Benchmarks {

BENCHMARK_DEFINE_F(GeneralFixture, ApproxModReduction)(benchmark::State& state) {
    if (this->generalTestParams.multDepth <= static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L <= level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);

    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};
    std::vector<double> x2 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));
    lbcrypto::Plaintext ptxt2 = cc->MakeCKKSPackedPlaintext(x2, 1, state.range(3));

    ptxt1->SetLevel(state.range(3));
    ptxt2->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);
    auto c2 = cc->Encrypt(keys.publicKey, ptxt2);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    FIDESlib::CKKS::RawCipherText raw2 = FIDESlib::CKKS::GetRawCipherText(cc, c2);

    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);
    FIDESlib::CKKS::Ciphertext GPUct2(GPUcc, raw2);

    FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
    kskEval.Initialize(GPUcc, rawKskEval);

    for (auto _ : state) {
        CudaCheckErrorMod;
        auto start = std::chrono::high_resolution_clock::now();
        FIDESlib::CKKS::approxModReduction(GPUct1, GPUct2, kskEval, 1.0);
        CudaCheckErrorMod;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
        state.SetIterationTime(elapsed.count());

        GPUct1.c0.grow(GPUcc.L - state.range(3));
        GPUct1.c1.grow(GPUcc.L - state.range(3));
        GPUct2.c0.grow(GPUcc.L - state.range(3));
        GPUct2.c1.grow(GPUcc.L - state.range(3));
    }
    CudaCheckErrorMod;
    cc->GetEvalAutomorphismKeyMap(this->keys.publicKey->GetKeyTag()).clear();
}

BENCHMARK_DEFINE_F(GeneralFixture, ApproxModReductionSparse)(benchmark::State& state) {
    if (this->generalTestParams.multDepth <= static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L <= level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);
    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));

    ptxt1->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);

    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
    kskEval.Initialize(GPUcc, rawKskEval);

    for (auto _ : state) {
        CudaCheckErrorMod;
        auto start = std::chrono::high_resolution_clock::now();
        FIDESlib::CKKS::approxModReductionSparse(GPUct1, kskEval, 1.0);
        CudaCheckErrorMod;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
        state.SetIterationTime(elapsed.count());

        GPUct1.c0.grow(GPUcc.L - state.range(3));
        GPUct1.c1.grow(GPUcc.L - state.range(3));
    }
    CudaCheckErrorMod;
    cc->GetEvalAutomorphismKeyMap(this->keys.publicKey->GetKeyTag()).clear();
}

BENCHMARK_DEFINE_F(GeneralFixture, CoeffsToSlots)(benchmark::State& state) {
    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    state.counters["p_batch"] = state.range(2);
    state.counters["p_slots"] = state.range(3);
    fideslibParams.batch = state.range(2);
    const int slots = state.range(3);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};
    std::vector<double> x2 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    cc->EvalBootstrapSetup({2, 2}, {0, 0}, slots);
    cc->EvalBootstrapKeyGen(keys.secretKey, slots);

    FIDESlib::CKKS::AddBootstrapPrecomputation(cc, keys, slots, GPUcc);
    const int start_level = GPUcc.GetBootPrecomputation(slots).CtS.at(0).A.at(0).c0.getLevel();
    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, GPUcc.L - start_level);

    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);

    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    {
        FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
        FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
        kskEval.Initialize(GPUcc, rawKskEval);
        GPUcc.AddEvalKey(std::move(kskEval));
    }

    for (auto _ : state) {
        CudaCheckErrorMod;
        auto start = std::chrono::high_resolution_clock::now();

        FIDESlib::CKKS::EvalCoeffsToSlots(GPUct1, slots, false);
        CudaCheckErrorMod;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
        state.SetIterationTime(elapsed.count());

        GPUct1.c0.grow(start_level);
        GPUct1.c1.grow(start_level);
    }
    CudaCheckErrorMod;
    cc->GetEvalAutomorphismKeyMap(this->keys.publicKey->GetKeyTag()).clear();
}

BENCHMARK_DEFINE_F(GeneralFixture, SlotsToCoeffs)(benchmark::State& state) {
    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    state.counters["p_batch"] = state.range(2);
    state.counters["p_slots"] = state.range(3);

    fideslibParams.batch = state.range(2);
    const int slots = state.range(3);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};
    std::vector<double> x2 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    cc->EvalBootstrapSetup({2, 2}, {0, 0}, slots);
    cc->EvalBootstrapKeyGen(keys.secretKey, slots);

    FIDESlib::CKKS::AddBootstrapPrecomputation(cc, keys, slots, GPUcc);

    const int init_level = GPUcc.GetBootPrecomputation(slots).StC.at(0).A.at(0).c0.getLevel();
    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, GPUcc.L - init_level);
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);

    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    {
        FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
        FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
        kskEval.Initialize(GPUcc, rawKskEval);
        GPUcc.AddEvalKey(std::move(kskEval));
    }

    for (auto _ : state) {
        auto start = std::chrono::high_resolution_clock::now();

        FIDESlib::CKKS::EvalCoeffsToSlots(GPUct1, slots, true);
        CudaCheckErrorMod;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
        state.SetIterationTime(elapsed.count());

        GPUct1.c0.grow(init_level);
        GPUct1.c1.grow(init_level);
    }
    CudaCheckErrorMod;
    cc->GetEvalAutomorphismKeyMap(this->keys.publicKey->GetKeyTag()).clear();
}

struct BootConfig {
    uint32_t slots, a, b, dim1, dim2;
};

BootConfig conf[] = {BootConfig{1 << 6, 1, 1, 16, 16},  BootConfig{1 << 9, 2, 2, 16, 16},
                     BootConfig{1 << 14, 5, 5, 8, 8},   BootConfig{1 << 15, 5, 5, 8, 8},
                     BootConfig{1 << 14, 4, 4, 8, 8},   BootConfig{1 << 15, 4, 4, 8, 8},
                     BootConfig{1 << 14, 3, 3, 16, 16}, BootConfig{1 << 15, 3, 3, 16, 16},
                     BootConfig{1 << 16, 4, 4, 16, 16}, BootConfig{1 << 16, 3, 3, 16, 16}};

#include <openfhe/pke/openfhe.h>

BENCHMARK_DEFINE_F(GeneralFixture, BootstrapGPU)(benchmark::State& state) {
    CudaCheckErrorMod;
    std::vector<int> GPUs = generalTestParams.GPUs;

    state.counters["p_batch"] = state.range(2);
    state.counters["p_slots"] = conf[state.range(3)].slots;

    fideslibParams.batch = state.range(2);
    const int slots = conf[state.range(3)].slots;
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    hipSetDevice(GPUs[0]);
    CudaCheckErrorMod;
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};
    std::vector<double> x2 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1);

    cc->EvalBootstrapSetup({conf[state.range(3)].a, conf[state.range(3)].b},
                           {conf[state.range(3)].dim1, conf[state.range(3)].dim2}, slots);
    cc->EvalBootstrapKeyGen(keys.secretKey, slots);

    FIDESlib::CKKS::AddBootstrapPrecomputation(cc, keys, slots, GPUcc);

    const int init_level = 1;
    ptxt1->SetLevel(GPUcc.L - init_level);
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    std::cout << "LoadEvalKey" << std::endl;
    {
        FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
        FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
        kskEval.Initialize(GPUcc, rawKskEval);
        GPUcc.AddEvalKey(std::move(kskEval));
    }

    int endlevel = 0;

    for (auto _ : state) {
        // std::cout << "Boot" << std::endl;
        hipDeviceSynchronize();
        auto start = std::chrono::high_resolution_clock::now();

        FIDESlib::CKKS::Bootstrap(GPUct1, slots, false);
        CudaCheckErrorMod;
        // std::cout << "end1" << std::endl;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
        state.SetIterationTime(elapsed.count());
        endlevel = GPUct1.getLevel();
        GPUct1.c0.grow(init_level + 1);
        GPUct1.c1.grow(init_level + 1);
        //  std::cout << "end" << std::endl;
    }
    std::cout << "Remaining levels: " << endlevel << std::endl;
    CudaCheckErrorMod;
    cc->GetEvalAutomorphismKeyMap(this->keys.publicKey->GetKeyTag()).clear();
}

BENCHMARK_REGISTER_F(GeneralFixture, ApproxModReduction)
    ->ArgsProduct({PARAMETERS, {0}, BATCH_CONFIG, {0, 1, 2, 3, 4, 5}})
    ->Iterations(50);
BENCHMARK_REGISTER_F(GeneralFixture, ApproxModReductionSparse)
    ->ArgsProduct({PARAMETERS, {0}, BATCH_CONFIG, {0, 1, 2, 3, 4, 5}})
    ->Iterations(50);
BENCHMARK_REGISTER_F(GeneralFixture, CoeffsToSlots)->ArgsProduct({{3}, {0}, BATCH_CONFIG, {64}})->Iterations(50);
BENCHMARK_REGISTER_F(GeneralFixture, SlotsToCoeffs)->ArgsProduct({{3}, {0}, BATCH_CONFIG, {64}})->Iterations(50);

BENCHMARK_REGISTER_F(GeneralFixture, BootstrapGPU)
    ->ArgsProduct({{3, 4}, {0}, BATCH_CONFIG, {0, 1, 2, 3, 4, 5, 6, 7}})
    ->Iterations(50);

BENCHMARK_REGISTER_F(GeneralFixture, BootstrapGPU)
    ->ArgsProduct({{6}, {0}, BATCH_CONFIG, {0, 1, 3, 5, 7, 8, 9}})
    ->Iterations(50);

BENCHMARK_DEFINE_F(GeneralFixture, BootstrapCPU)(benchmark::State& state) {

    int devcount = -1;
    hipGetDeviceCount(&devcount);

    std::vector<int> GPUs = generalTestParams.GPUs;

    state.counters["p_batch"] = state.range(2);
    state.counters["p_slots"] = conf[state.range(3)].slots;

    fideslibParams.batch = state.range(2);
    const int slots = conf[state.range(3)].slots;
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};
    std::vector<double> x2 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, GPUcc.L - 1, nullptr, slots);

    cc->EvalBootstrapSetup({conf[state.range(3)].a, conf[state.range(3)].b}, {0, 0}, slots);
    cc->EvalBootstrapKeyGen(keys.secretKey, slots);

    //FIDESlib::CKKS::AddBootstrapPrecomputation(cc, keys, slots, GPUcc);

    //const int init_level = GPUcc.GetBootPrecomputation(slots).StC.at(0).A.at(0).c0.getLevel();
    ////ptxt1->SetLevel(GPUcc.L - init_level);
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    //FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    //FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    {
        //FIDESlib::CKKS::KeySwitchingKey kskEval(GPUcc);
        //FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetEvalKeySwitchKey(keys);
        //kskEval.Initialize(GPUcc, rawKskEval);
        // GPUcc.AddEvalKey(std::move(kskEval));
    }

    int endlevel = 0;

    for (auto _ : state) {
        auto ct = c1->Clone();

        auto start = std::chrono::high_resolution_clock::now();
        ct = cc->EvalBootstrap(ct);
        //FIDESlib::CKKS::Bootstrap(GPUct1, slots);
        CudaCheckErrorMod;
        auto end = std::chrono::high_resolution_clock::now();
        auto elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end - start);
        state.SetIterationTime(elapsed.count());
    }
    std::cout << "Remaining levels: " << endlevel << std::endl;
    CudaCheckErrorMod;
    cc->GetEvalAutomorphismKeyMap(this->keys.publicKey->GetKeyTag()).clear();
}
BENCHMARK_REGISTER_F(GeneralFixture, BootstrapCPU)->ArgsProduct({{3}, {0}, {1}, {0, 1, 6, 7}})->Iterations(3);

//BENCHMARK_REGISTER_F(GeneralFixture, BootstrapCPU)->ArgsProduct({{4, 3}, {0}, {2, 6, 12}, {4, 5}});

}  // namespace FIDESlib::Benchmarks
