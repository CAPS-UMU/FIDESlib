//
// Created by carlosad on 5/11/24.
//
#include <benchmark/benchmark.h>

#include "Benchmark.cuh"
#include "CKKS/AccumulateBroadcast.cuh"
#include "CKKS/KeySwitchingKey.cuh"
#include "CKKS/openfhe-interface/RawCiphertext.cuh"

namespace FIDESlib::Benchmarks {
BENCHMARK_DEFINE_F(GeneralFixture, CiphertextRotation)(benchmark::State& state) {
    if (this->generalTestParams.multDepth < static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L < level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);
    std::vector<int> GPUs = generalTestParams.GPUs;

    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));
    ptxt1->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);

    FIDESlib::CKKS::KeySwitchingKey kskRot(GPUcc);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval = FIDESlib::CKKS::GetRotationKeySwitchKey(keys, 1, cc);
    kskRot.Initialize(GPUcc, rawKskEval);

    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);
    for (auto _ : state) {
        GPUct1.rotate(2 * GPUcc.N - 1, kskRot, true);
        CudaCheckErrorMod;
    }
    CudaCheckErrorMod;
}

BENCHMARK_DEFINE_F(GeneralFixture, CiphertextHoistedRotation)(benchmark::State& state) {
    if (this->generalTestParams.multDepth < static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L < level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);
    std::vector<int> GPUs = generalTestParams.GPUs;

    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));
    ptxt1->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);
    FIDESlib::CKKS::Ciphertext GPUct2(GPUcc, raw1);
    FIDESlib::CKKS::Ciphertext GPUct3(GPUcc, raw1);
    FIDESlib::CKKS::Ciphertext GPUct4(GPUcc, raw1);

    FIDESlib::CKKS::KeySwitchingKey kskRot1(GPUcc);
    FIDESlib::CKKS::KeySwitchingKey kskRot2(GPUcc);
    FIDESlib::CKKS::KeySwitchingKey kskRot3(GPUcc);
    FIDESlib::CKKS::KeySwitchingKey kskRot4(GPUcc);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval1 = FIDESlib::CKKS::GetRotationKeySwitchKey(keys, 1, cc);
    kskRot1.Initialize(GPUcc, rawKskEval1);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval2 = FIDESlib::CKKS::GetRotationKeySwitchKey(keys, 2, cc);
    kskRot2.Initialize(GPUcc, rawKskEval2);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval3 = FIDESlib::CKKS::GetRotationKeySwitchKey(keys, 3, cc);
    kskRot3.Initialize(GPUcc, rawKskEval3);
    FIDESlib::CKKS::RawKeySwitchKey rawKskEval4 = FIDESlib::CKKS::GetRotationKeySwitchKey(keys, 4, cc);
    kskRot4.Initialize(GPUcc, rawKskEval4);

    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);
    for (auto _ : state) {
        GPUct1.rotate_hoisted({&kskRot1, &kskRot2, &kskRot3, &kskRot4}, {1, 2, 3, 4},
                              {&GPUct2, &GPUct3, &GPUct4, &GPUct1}, false);
        CudaCheckErrorMod;
    }
    CudaCheckErrorMod;
}

BENCHMARK_DEFINE_F(GeneralFixture, CiphertextRotateAndAccumulate)(benchmark::State& state) {
    if (this->generalTestParams.multDepth < static_cast<uint64_t>(state.range(3))) {
        state.SkipWithMessage("cc.L < level");
        return;
    }

    int devcount = -1;
    hipGetDeviceCount(&devcount);
    std::vector<int> GPUs = generalTestParams.GPUs;

    fideslibParams.batch = state.range(2);
    FIDESlib::CKKS::RawParams raw_param = FIDESlib::CKKS::GetRawParams(cc);
    FIDESlib::CKKS::Context GPUcc{fideslibParams.adaptTo(raw_param), GPUs};

    std::vector<double> x1 = {0.25, 0.5, 0.75, 1.0, 2.0, 3.0, 4.0, 5.0};

    lbcrypto::Plaintext ptxt1 = cc->MakeCKKSPackedPlaintext(x1, 1, state.range(3));
    ptxt1->SetLevel(state.range(3));
    auto c1 = cc->Encrypt(keys.publicKey, ptxt1);

    int bstep = state.range(4);
    std::vector<int> indexes = FIDESlib::CKKS::GetAccumulateRotationIndices(bstep, 1, GPUcc.N / 2);
    FIDESlib::CKKS::GenAndAddRotationKeys(cc, keys, GPUcc, indexes);

    FIDESlib::CKKS::RawCipherText raw1 = FIDESlib::CKKS::GetRawCipherText(cc, c1);
    FIDESlib::CKKS::Ciphertext GPUct1(GPUcc, raw1);
    state.counters["p_batch"] = state.range(2);
    state.counters["p_limbs"] = state.range(3);
    for (auto _ : state) {

        FIDESlib::CKKS::Accumulate(GPUct1, bstep, 1, GPUcc.N / 2);
        CudaCheckErrorMod;
    }
    CudaCheckErrorMod;
}

BENCHMARK_REGISTER_F(GeneralFixture, CiphertextRotation)->ArgsProduct({PARAMETERS, {0}, BATCH_CONFIG, LEVEL_CONFIG});
BENCHMARK_REGISTER_F(GeneralFixture, CiphertextHoistedRotation)
    ->ArgsProduct({PARAMETERS, {0}, BATCH_CONFIG, LEVEL_CONFIG});

BENCHMARK_REGISTER_F(GeneralFixture, CiphertextRotateAndAccumulate)
    ->ArgsProduct({PARAMETERS, {0}, BATCH_CONFIG, LEVEL_CONFIG, {2, 4, 8}});
}  // namespace FIDESlib::Benchmarks