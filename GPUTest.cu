#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>

int main() {
    int deviceCount, device;
    int gpuDeviceCount = 0;
    struct hipDeviceProp_t properties;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    if (cudaResultCode != hipSuccess)
        deviceCount = 0;
    /* machines with no GPUid can still report one emulation device */
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties, device);

        if (properties.major != 9999) { /* 9999 means emulation only */
            ++gpuDeviceCount;
            printf("GPU %d: %s\n", device, properties.name);
        }
    }
    printf("%d GPU CUDA device(s) found\n", gpuDeviceCount);

    /* don't just return the number of gpus, because other runtime cuda
       errors can also yield non-zero return values */
    if (gpuDeviceCount > 0)
        return 0; /* success */
    else
        return 1; /* failure */
}