#include "hip/hip_runtime.h"
//
// Created by carlosad on 2/05/24.
//
#include "CKKS/Context.cuh"

#include <source_location>
#include "CKKS/BootstrapPrecomputation.cuh"
#include "CKKS/Ciphertext.cuh"
#include "CKKS/KeySwitchingKey.cuh"

namespace FIDESlib::CKKS {

constexpr bool SPLIT_SPECIAL = true;

Context::Context(Parameters param, const std::vector<int>& devs, const int secBits)
    : my_range(loc, LIFETIME),
      param((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))),
             param)),
      logN(param.logN),
      N(1 << logN),
      slots(1 << (logN - 1)),
      rescaleTechnique(translateRescalingTechnique(param.scalingTechnique)),
      L(param.L),
      logQ(computeLogQ(L, param.primes)),
      batch(param.batch),
      GPUid(devs),
      dnum(validateDnum(GPUid, param.dnum)),
      GPUdigits(generateGPUdigits(dnum, devs)),
      prime(param.primes.begin(), param.primes.begin() + L + 1),
      meta{generateMeta(GPUid, dnum, GPUdigits, prime, param)},
      logQ_d(computeLogQ_d(dnum, meta, prime)),
      K(computeK(logQ_d, param.Sprimes, param)),
      logP(computeLogQ(K - 1, param.Sprimes)),
      specialPrime(param.Sprimes.begin(), param.Sprimes.begin() + K),
      specialMeta(generateSpecialMeta(meta, specialPrime, L + 1, GPUid)),
      splitSpecialMeta(generateSplitSpecialMeta(specialMeta.at(0), GPUid)),
      decompMeta(generateDecompMeta(meta, GPUdigits, GPUid, L)),
      digitMeta(generateDigitMeta(meta, splitSpecialMeta, specialMeta.at(0), GPUdigits, GPUid)),
      gatherMeta(generateGatherMeta(meta, L)),
      limbGPUid(generateLimbGPUid(meta, L)),
      digitGPUid(generateDigitGPUid(meta, L, dnum)),
      GPUrank(GPUid.size())
//top_limb(devs.size())
{
    SetupConstants<Parameters>(prime, meta, specialPrime, specialMeta.at(0), decompMeta, digitMeta, GPUdigits, GPUid, N,
                               param);
#ifdef NCCL
    PrepareNCCLCommunication();
#endif
    // CheckBitSecurity();
    int bits = 0;
    for (auto& j : {prime, specialPrime})
        for (auto& i : j)
            bits += i.bits;

    std::cout << "bits: (slightly overestimated) " << bits << std::endl;

    for (int dev : GPUid) {
        hipSetDevice(dev);
        hipMemPool_t mp;
        hipDeviceGetDefaultMemPool(&mp, dev);
        uint64_t threshold = UINT64_MAX;  //5l * 1024l * 1024l * 1024l;  // One Gigabyte of memory
        hipMemPoolSetAttribute(mp, hipMemPoolAttrReleaseThreshold, &threshold);
        CudaCheckErrorModNoSync;
    }

    CudaNvtxStop();
}

std::vector<dim3> Context::generateLimbGPUid(const std::vector<std::vector<LimbRecord>>& meta, const int L) {
    std::vector<dim3> res(L + 1, 0);
    for (int i = 0; i < static_cast<int>(meta.size()); ++i) {
        for (size_t j = 0; j < meta.at(i).size(); ++j) {
            res.at(meta[i][j].id) = {static_cast<uint32_t>(i), static_cast<uint32_t>(j), 0};
        }
    }
    return res;
}

std::vector<std::vector<std::vector<LimbRecord>>> Context::generateDigitMeta(
    const std::vector<std::vector<LimbRecord>>& meta, const std::vector<std::vector<LimbRecord>>& splitSpecialMeta,
    const std::vector<LimbRecord>& specialMeta, const std::vector<std::vector<int>>& digitGPUid,
    const std::vector<int>& GPUid) {
    std::vector<std::vector<std::vector<LimbRecord>>> digitMeta(meta.size());

    for (size_t i = 0; i < digitGPUid.size(); ++i) {
        hipSetDevice(GPUid[i]);
        for (int d : digitGPUid.at(i)) {
            digitMeta[i].emplace_back();

            if constexpr (SPLIT_SPECIAL) {
                for (auto& l : splitSpecialMeta.at(i)) {
                    digitMeta[i].back().emplace_back(LimbRecord{.id = l.id, .type = l.type, .digit = l.digit});
                    digitMeta[i].back().back().stream.init();
                }
            } else {
                for (auto& l : specialMeta) {
                    digitMeta[i].back().emplace_back(LimbRecord{.id = l.id, .type = l.type, .digit = l.digit});
                    digitMeta[i].back().back().stream.init();
                }
            }

            for (auto& l : meta.at(i)) {
                if (l.digit != d) {
                    digitMeta[i].back().emplace_back(LimbRecord{.id = l.id, .type = l.type, .digit = l.digit});
                    digitMeta[i].back().back().stream.init();
                }
            }

            /*
            std::sort(
                digitMeta[i].back().begin() + specialMeta.size(), digitMeta[i].back().end(),
                [](LimbRecord& a, LimbRecord& b) { return a.digit < b.digit || (a.digit == b.digit && a.id < b.id); });
            */
        }
    }
    return digitMeta;
}

std::vector<std::vector<std::vector<LimbRecord>>> Context::generateDecompMeta(
    const std::vector<std::vector<LimbRecord>>& meta, const std::vector<std::vector<int>> digitGPUid,
    const std::vector<int>& GPUid, int L) {
    std::vector<std::vector<std::vector<LimbRecord>>> decompMeta(meta.size());

    for (size_t i = 0; i < digitGPUid.size(); ++i) {
        hipSetDevice(GPUid[i]);
        for (int d : digitGPUid.at(i)) {
            decompMeta[i].emplace_back();

            for (int primeid = 0; primeid <= L; ++primeid) {
                for (auto& m : meta) {
                    for (auto& l : m) {
                        if (l.id == primeid && l.digit == d) {
                            decompMeta[i].back().push_back(LimbRecord{.id = l.id, .type = l.type, .digit = l.digit});
                            decompMeta[i].back().back().stream.init();
                        }
                    }
                }
            }
        }
    }

    return decompMeta;
}

bool Context::isValidPrimeId(const int i) const {
    return (i >= 0 && i < L + 1 + K);
}

int Context::computeLogQ(const int L, std::vector<PrimeRecord>& primes) {
    int res = 0;
    assert(L <= (int)primes.size());
    for (int i = 0; i <= L; ++i) {
        res += (primes[i].bits == -1) ? (primes[i].bits = (int)std::bit_width(primes[i].p)) : primes[i].bits;
    }
    return res;
}

int Context::validateDnum(const std::vector<int>& GPUid, const int dnum) {
    return dnum;
}

int findDigitOnParam(const Parameters& param, uint64_t modulus) {
    for (size_t i = 0; i < param.raw->PARTITIONmoduli.size(); ++i) {
        for (uint64_t j : param.raw->PARTITIONmoduli.at(i)) {
            if (modulus == j)
                return i;
        }
    }
    return -1;
}

std::vector<std::vector<LimbRecord>> Context::generateMeta(const std::vector<int>& GPUid, const int dnum,
                                                           const std::vector<std::vector<int>> digitGPUid,
                                                           const std::vector<PrimeRecord>& prime,
                                                           const Parameters& param) {
    int devs = GPUid.size();
    std::vector<std::vector<LimbRecord>> meta(devs);

    //for (int i = 0; i < devs; ++i) {
    // hipSetDevice(GPUid.at(i));
    // meta.at(i).resize((prime.size() + devs - i - 1) / devs);
    //}

    if constexpr (0) {
        int threshhold1 = (prime.size() / 2 + devs - 1) / devs;
        int threshhold2 = threshhold1 * devs;
        int dev = 0;
        for (int i = 0; i < (int)prime.size(); ++i) {
            int digit_ = !param.raw ? i % dnum : findDigitOnParam(param, prime.at(i).p);

            if (i < threshhold1) {
            } else if (i < threshhold2) {
                dev = (dev + 1) % devs;
                if (dev == 0)
                    dev = (dev + 1) % devs;
            } else {
                dev = (dev + 1) % devs;
            }
            /*{
            int dev = -1;
            for (size_t j = 0; j < digitGPUid.size(); ++j) {
                for (auto& k : digitGPUid.at(j))
                    if (k == digit)
                        dev = j;
            }
        }*/

            hipSetDevice(GPUid[dev]);

            meta[dev].push_back(
                LimbRecord{.id = i,
                           .type = (prime[i].type ? *(prime[i].type) : (prime[i].bits <= 30 ? U32 : U64)),
                           .digit = digit_});
            meta[dev].back().stream.init();
            // std::cout << "i: " << i << " gpu:" << dev << std::endl;
        }
    } else {
        for (int i = 0; i < (int)prime.size(); ++i) {
            int digit_ = !param.raw ? i % dnum : findDigitOnParam(param, prime.at(i).p);

            int dev = i % GPUid.size();
            /*{
            int dev = -1;
            for (size_t j = 0; j < digitGPUid.size(); ++j) {
                for (auto& k : digitGPUid.at(j))
                    if (k == digit)
                        dev = j;
            }
        }*/

            hipSetDevice(GPUid[dev]);

            meta[dev].push_back(
                LimbRecord{.id = i,
                           .type = (prime[i].type ? *(prime[i].type) : (prime[i].bits <= 30 ? U32 : U64)),
                           .digit = digit_});
            meta[dev].back().stream.init();
        }
    }

    return meta;
}

std::vector<int> Context::computeLogQ_d(const int dnum, const std::vector<std::vector<LimbRecord>>& meta,
                                        const std::vector<PrimeRecord>& prime) {
    std::vector<int> logQ_d(dnum, 0);

    for (auto& i : meta)
        for (auto& j : i)
            logQ_d.at(j.digit) += prime.at(j.id).bits;

    return logQ_d;
}

int Context::computeK(const std::vector<int>& logQ_d, std::vector<PrimeRecord>& Sprimes, const Parameters& param) {

    size_t res = 0;
    int logMaxD = *std::max_element(logQ_d.begin(), logQ_d.end());
    int bits = 0;
    for (; bits < logMaxD && res < Sprimes.size(); ++res) {
        bits += (Sprimes.at(res).bits <= 0) ? (Sprimes.at(res).bits = (int)std::bit_width(Sprimes.at(res).p)) - 1
                                            : Sprimes.at(res).bits - 1;
    }

    if (param.K != -1) {
        return param.K;
    }
    assert(bits >= logMaxD);
    return res;
}

std::vector<std::vector<LimbRecord>> Context::generateSpecialMeta(const std::vector<std::vector<LimbRecord>>& meta,
                                                                  const std::vector<PrimeRecord>& specialPrime,
                                                                  const int ID0, const std::vector<int>& GPUid) {
    std::vector<std::vector<LimbRecord>> specialMeta(GPUid.size());

    for (size_t d = 0; d < GPUid.size(); ++d) {
        specialMeta.at(d).resize(specialPrime.size());
        hipSetDevice(GPUid[d]);
        for (int i = 0; i < (int)specialPrime.size(); ++i) {
            specialMeta.at(d).at(i).id = ID0 + i;
            specialMeta.at(d).at(i).type =
                (specialPrime[i].type ? *(specialPrime[i].type) : (specialPrime[i].bits <= 30 ? U32 : U64));
            specialMeta.at(d).at(i).stream.init();
        }
    }

    return specialMeta;
}

std::vector<LimbRecord> Context::generateGatherMeta(const std::vector<std::vector<LimbRecord>>& meta, int L) {
    std::vector<LimbRecord> gatherMeta(L + 1);

    /*TODO: generate streams for each device*/
    for (int i = 0; i <= L; ++i) {
        for (size_t j = 0; j < meta.size(); ++j) {
            for (size_t k = 0; k < meta.at(j).size(); ++k) {
                if (meta[j][k].id == i) {
                    gatherMeta.at(i).id = i;
                    gatherMeta.at(i).digit = meta[j][k].digit;
                    gatherMeta.at(i).type = meta[j][k].type;
                }
            }
        }
    }

    return gatherMeta;
}

std::vector<std::vector<int>> Context::generateGPUdigits(const int dnum, const std::vector<int>& devs) {
    std::vector<std::vector<int>> res(devs.size());
    for (int d = 0; d < dnum; ++d) {
        for (int gpu = 0; gpu < devs.size(); ++gpu) {
            res[gpu].push_back(d);
        }
    }
    return res;
}

RNSPoly& Context::getKeySwitchAux() {
    if (key_switch_aux == nullptr)
        key_switch_aux = std::make_unique<RNSPoly>(*this, L, true);
    key_switch_aux->generateDecompAndDigit(false);
    key_switch_aux->generateSpecialLimbs(false);
    return *key_switch_aux;
}

RNSPoly& Context::getKeySwitchAux2() {
    if (key_switch_aux2 == nullptr)
        key_switch_aux2 = std::make_unique<RNSPoly>(*this, L, true);
    key_switch_aux2->generateDecompAndDigit(false);
    key_switch_aux2->generateSpecialLimbs(false);
    return *key_switch_aux2;
}

RNSPoly& Context::getModdownAux(const int num) {
    if (moddown_aux[num % moddown_aux.size()] == nullptr)
        moddown_aux[num % moddown_aux.size()] = std::make_unique<RNSPoly>(*this, L, true);
    return *moddown_aux[num % moddown_aux.size()];
}
std::vector<uint64_t> Context::ElemForEvalMult(int level, const double operand) {

    uint32_t numTowers = level + 1;
    std::vector<lbcrypto::DCRTPoly::Integer> moduli(numTowers);
    for (usint i = 0; i < numTowers; i++) {
        moduli[i] = prime[i].p;
    }

    double scFactor = param.ScalingFactorReal[level];

    typedef int128_t DoubleInteger;
    int32_t MAX_BITS_IN_WORD_LOCAL = 125;

    int32_t logApprox = 0;
    const double res = std::fabs(operand * scFactor);
    if (res > 0) {
        int32_t logSF = static_cast<int32_t>(std::ceil(std::log2(res)));
        int32_t logValid = (logSF <= MAX_BITS_IN_WORD_LOCAL) ? logSF : MAX_BITS_IN_WORD_LOCAL;
        logApprox = logSF - logValid;
    }
    double approxFactor = pow(2, logApprox);

    DoubleInteger large = static_cast<DoubleInteger>(operand / approxFactor * scFactor + 0.5);
    DoubleInteger large_abs = (large < 0 ? -large : large);
    DoubleInteger bound = (uint64_t)1 << 63;

    std::vector<lbcrypto::DCRTPoly::Integer> factors(numTowers);

    if (large_abs > bound) {
        for (usint i = 0; i < numTowers; i++) {
            DoubleInteger reduced = large % moduli[i].ConvertToInt();

            factors[i] = (reduced < 0) ? static_cast<uint64_t>(reduced + moduli[i].ConvertToInt())
                                       : static_cast<uint64_t>(reduced);
        }
    } else {
        int64_t scConstant = static_cast<int64_t>(large);
        for (usint i = 0; i < numTowers; i++) {
            int64_t reduced = scConstant % static_cast<int64_t>(moduli[i].ConvertToInt());

            factors[i] = (reduced < 0) ? reduced + moduli[i].ConvertToInt() : reduced;
        }
    }

    // Scale back up by approxFactor within the CRT multiplications.
    if (logApprox > 0) {
        int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                              ? logApprox
                              : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
        lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
        std::vector<lbcrypto::DCRTPoly::Integer> crtApprox(numTowers, intStep);
        logApprox -= logStep;

        while (logApprox > 0) {
            int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                                  ? logApprox
                                  : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
            lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
            std::vector<lbcrypto::DCRTPoly::Integer> crtSF(numTowers, intStep);
            crtApprox = lbcrypto::CKKSPackedEncoding::CRTMult(crtApprox, crtSF, moduli);
            logApprox -= logStep;
        }
        factors = lbcrypto::CKKSPackedEncoding::CRTMult(factors, crtApprox, moduli);
    }

    std::vector<uint64_t> result(numTowers);
    for (int i = 0; i < result.size(); ++i) {
        result[i] = factors[i].ConvertToInt();
        result[i] = result[i] % prime[i].p;
    }

    return result;
}

std::ostream& operator<<(std::ostream& o, const uint128_t& x) {
    if (x == std::numeric_limits<uint128_t>::min())
        return o << "0";
    if (x < 10)
        return o << (char)(x + '0');
    return o << x / 10 << (char)(x % 10 + '0');
}

std::vector<uint64_t> Context::ElemForEvalAddOrSub(const int level, const double operand, const int noise_deg) {
    usint sizeQl = level + 1;
    std::vector<lbcrypto::DCRTPoly::Integer> moduli(sizeQl);
    for (usint i = 0; i < sizeQl; i++) {
        moduli[i] = prime[i].p;
    }

    //double scFactor = param.ScalingFactorReal.at(level);
    double scFactor = 0;
    if (this->rescaleTechnique == FLEXIBLEAUTOEXT && level == L) {
        scFactor =
            param.ScalingFactorRealBig.at(level);  // cryptoParams->GetScalingFactorRealBig(ciphertext->GetLevel());
    } else {
        scFactor = param.ScalingFactorReal.at(level);  //cryptoParams->GetScalingFactorReal(ciphertext->GetLevel());
    }

    int32_t logApprox = 0;
    const double res = std::fabs(operand * scFactor);
    if (res > 0) {
        int32_t logSF = static_cast<int32_t>(std::ceil(std::log2(res)));
        int32_t logValid = (logSF <= lbcrypto::LargeScalingFactorConstants::MAX_BITS_IN_WORD)
                               ? logSF
                               : lbcrypto::LargeScalingFactorConstants::MAX_BITS_IN_WORD;
        logApprox = logSF - logValid;
    }
    double approxFactor = pow(2, logApprox);

    lbcrypto::DCRTPoly::Integer scConstant = static_cast<uint64_t>(operand * scFactor / approxFactor + 0.5);
    std::vector<lbcrypto::DCRTPoly::Integer> crtConstant(sizeQl, scConstant);

    // Scale back up by approxFactor within the CRT multiplications.
    if (logApprox > 0) {
        int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                              ? logApprox
                              : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
        lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
        std::vector<lbcrypto::DCRTPoly::Integer> crtApprox(sizeQl, intStep);
        logApprox -= logStep;

        while (logApprox > 0) {
            int32_t logStep = (logApprox <= lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP)
                                  ? logApprox
                                  : lbcrypto::LargeScalingFactorConstants::MAX_LOG_STEP;
            lbcrypto::DCRTPoly::Integer intStep = uint64_t(1) << logStep;
            std::vector<lbcrypto::DCRTPoly::Integer> crtSF(sizeQl, intStep);
            crtApprox = lbcrypto::CKKSPackedEncoding::CRTMult(crtApprox, crtSF, moduli);
            logApprox -= logStep;
        }
        crtConstant = lbcrypto::CKKSPackedEncoding::CRTMult(crtConstant, crtApprox, moduli);
    }

    // In FLEXIBLEAUTOEXT mode at level 0, we don't use the depth to calculate the scaling factor,
    // so we return the value before taking the depth into account.
    if (this->rescaleTechnique == FLEXIBLEAUTOEXT && level == L) {
        std::vector<uint128_t> result(sizeQl);
        for (int i = 0; i < result.size(); ++i) {
            result[i] = crtConstant[i].ConvertToInt<uint128_t>();
        }

        for (int i = 0; i < result.size(); ++i) {
            result[i] = result[i] % prime[i].p;
        }

        std::vector<uint64_t> result2(crtConstant.size());
        for (int i = 0; i < result.size(); ++i) {
            result2[i] = result[i];
        }

        return result2;
    }

    lbcrypto::DCRTPoly::Integer intScFactor = static_cast<uint64_t>(scFactor + 0.5);
    std::vector<lbcrypto::DCRTPoly::Integer> crtScFactor(sizeQl, intScFactor);

    for (usint i = 1; i < noise_deg; i++) {
        crtConstant = lbcrypto::CKKSPackedEncoding::CRTMult(crtConstant, crtScFactor, moduli);
    }

    std::vector<uint128_t> result(sizeQl);
    for (int i = 0; i < result.size(); ++i) {
        result[i] = crtConstant[i].ConvertToInt<uint128_t>();
    }

    for (int i = 0; i < result.size(); ++i) {
        result[i] = result[i] % prime[i].p;
    }

    std::vector<uint64_t> result2(crtConstant.size());
    for (int i = 0; i < result.size(); ++i) {
        result2[i] = result[i];
    }

    return result2;
}
std::vector<double>& Context::GetCoeffsChebyshev() {
    assert(param.raw);
    return param.raw->coefficientsCheby;
}
int Context::GetDoubleAngleIts() {
    assert(param.raw);
    return param.raw ? param.raw->doubleAngleIts : 3;
}

int Context::GetBootK() {
    assert(param.raw);
    return param.raw ? param.raw->bootK : 1;
}

std::map<int, BootstrapPrecomputation> boot_precomps;

bool Context::HasBootPrecomputation(int slots) {
    return boot_precomps.contains(slots);
}
BootstrapPrecomputation& Context::GetBootPrecomputation(int slots) {
    if (!boot_precomps.contains(slots))
        assert("No precomputation." == nullptr);
    return boot_precomps[slots];
}

std::map<int, KeySwitchingKey> rot_keys;

KeySwitchingKey& Context::GetRotationKey(int index) {
    //index = index % (cc.N / 2);
    if (index < 0)
        index += this->N / 2;
    return rot_keys.at(index);
}
void Context::AddRotationKey(int index, KeySwitchingKey&& ksk) {
    //index = index % (cc.N / 2);
    if (index < 0)
        index += this->N / 2;
    rot_keys.emplace(index, std::move(ksk));
}
bool Context::HasRotationKey(int index) {
    //index = index % (cc.N / 2);
    if (index < 0)
        index += this->N / 2;
    return rot_keys.contains(index);
}

std::optional<KeySwitchingKey> eval_key;

void Context::AddEvalKey(KeySwitchingKey&& ksk) {
    eval_key.emplace(std::move(ksk));
}
KeySwitchingKey& Context::GetEvalKey() {
    return eval_key.value();
}

void Context::AddBootPrecomputation(int slots, BootstrapPrecomputation&& precomp) const {
    {
        std::cout << "Adding bootstrap precomputation to GPU for " << slots << " slots.\n"
                  << "Rotation keys loaded: " << rot_keys.size() << " ~ "
                  << 2 * ((long long)rot_keys.size() * dnum * (L + K + 1) * N * 8 / (1 << 20)) << "MB\n"
                  << "Plaintexts loaded: "
                  << (precomp.CtS.size() == 0 ? (precomp.LT.A.size() + precomp.LT.invA.size())
                                              : (precomp.StC.size() * precomp.StC.at(0).A.size() +
                                                 precomp.CtS.size() * precomp.CtS.at(0).A.size()))
                  << " ~ "
                  << (precomp.CtS.size() == 0
                          ? (precomp.LT.A.size() * (precomp.LT.A.at(0).c0.getLevel() +
                                                    precomp.LT.A.at(0).c0.isModUp() * specialMeta[0].size()) +
                             precomp.LT.invA.size() * (precomp.LT.invA.at(0).c0.getLevel() +
                                                       precomp.LT.invA.at(0).c0.isModUp() * specialMeta[0].size()))
                          : (precomp.StC.size() * precomp.StC.at(0).A.size() *
                                 (1 + precomp.StC.at(0).A.at(0).c0.getLevel() +
                                  precomp.StC.at(0).A.at(0).c0.isModUp() * specialMeta[0].size()) +
                             precomp.CtS.size() * precomp.CtS.at(0).A.size() *
                                 (1 + precomp.CtS.at(0).A.at(0).c0.getLevel() +
                                  precomp.CtS.at(0).A.at(0).c0.isModUp() * specialMeta[0].size()))) *
                         N * 8 / (1 << 20)
                  << "MB\n";
    }

    boot_precomps.emplace(slots, std::move(precomp));
}

Context::RESCALE_TECHNIQUE Context::translateRescalingTechnique(lbcrypto::ScalingTechnique technique) {
    return technique == lbcrypto::ScalingTechnique::FIXEDAUTO         ? Context::FIXEDAUTO
           : technique == lbcrypto::ScalingTechnique::FIXEDMANUAL     ? Context::FIXEDMANUAL
           : technique == lbcrypto::ScalingTechnique::FLEXIBLEAUTOEXT ? Context::FLEXIBLEAUTOEXT
           : technique == lbcrypto::ScalingTechnique::FLEXIBLEAUTO    ? Context::FLEXIBLEAUTO
                                                                      : Context::NO_RESCALE;
}

#ifdef NCCL
std::map<int, ncclComm_t*> devices;

void Context::PrepareNCCLCommunication() {

    if (GPUid.size() > 1) {
        NCCLCHECK(ncclGetUniqueId(&communicatorID));
        GPUrank.resize(GPUid.size());

        std::set<int> ids;
        for (int i : GPUid)
            ids.insert(i);
        int num_ranks = ids.size();

        bool p2p = true;
        for (auto& i : ids) {
            hipSetDevice(i);
            for (auto& j : ids) {
                if (i != j) {
                    int canAccessPeer;
                    hipDeviceCanAccessPeer(&canAccessPeer, i, j);
                    if (canAccessPeer)
                        hipDeviceEnablePeerAccess(j, 0);
                    else
                        p2p = false;

                    hipDeviceCanAccessPeer(&canAccessPeer, j, i);
                    if (canAccessPeer)
                        hipDeviceEnablePeerAccess(j, 0);
                    else
                        p2p = false;
                }
            }
        }
        this->canP2P = p2p;
        /*
        if (GPUid.size() > 1) {
            if (this->canP2P)
                std::cout << "P2P (Nvlink?) detected" << std::endl;
            else
                std::cout << "NO P2P" << std::endl;
        }*/

        ncclGroupStart();
        for (int i = 0; i < GPUid.size(); i++) {
            hipSetDevice(GPUid[i]);
            if (devices[GPUid[i]] == nullptr) {
                NCCLCHECK(ncclCommInitRank(GPUrank.data() + i, num_ranks, communicatorID, i));
                devices[GPUid[i]] = GPUrank.data() + i;
            } else {
                GPUrank[i] = *devices[GPUid[i]];
            }
        }
        ncclGroupEnd();

        hipDeviceSynchronize();

        top_limb_stream.resize(GPUid.size());
        top_limb_stream2.resize(GPUid.size());
        top_limb_buffer.resize(GPUid.size());
        top_limb_buffer2.resize(GPUid.size());
        top_limb_buffer_handle.resize(GPUid.size());
        top_limb_buffer2_handle.resize(GPUid.size());
        for (size_t i = 0; i < GPUid.size(); ++i) {
            hipSetDevice(GPUid[i]);

            top_limb_stream[i].init(100);
            top_limb_stream2[i].init(100);

            NCCLCHECK(ncclMemAlloc((void**)&top_limb_buffer[i], sizeof(uint64_t) * N));
            NCCLCHECK(
                ncclCommRegister(GPUrank[i], top_limb_buffer[i], sizeof(uint64_t) * N, &top_limb_buffer_handle[i]));
            NCCLCHECK(ncclMemAlloc((void**)&top_limb_buffer2[i], sizeof(uint64_t) * N));
            NCCLCHECK(
                ncclCommRegister(GPUrank[i], top_limb_buffer2[i], sizeof(uint64_t) * N, &top_limb_buffer2_handle[i]));

            hipDeviceSynchronize();
            top_limbptr.emplace_back(top_limb_stream[i], 1, GPUid[i], (void**)&top_limb_buffer[i]);
            top_limbptr2.emplace_back(top_limb_stream2[i], 1, GPUid[i], (void**)&top_limb_buffer2[i]);
        }

        gatherStream.resize(GPUid.size());
        for (size_t i = 0; i < GPUid.size(); ++i) {
            gatherStream[i].resize(GPUid.size());
            for (size_t j = 0; j < GPUid.size(); ++j) {
                hipSetDevice(GPUid[j]);
                gatherStream[i][j].init(100);
            }
        }

        /* for (int i = 0; i < dnum; ++i) {
            key_switch_digits.emplace_back(*this, L, true);
            key_switch_digits.back().generateSpecialLimbs();
        }*/

        CudaCheckErrorModNoSync;
    }

    digitStream.resize(dnum);
    for (int i = 0; i < dnum; ++i) {
        digitStream[i].resize(GPUid.size());
        for (size_t j = 0; j < GPUid.size(); ++j) {
            hipSetDevice(GPUid[j]);
            digitStream[i][j].init(100);
        }
    }
    digitStream2.resize(dnum);
    for (int i = 0; i < dnum; ++i) {
        digitStream2[i].resize(GPUid.size());
        for (size_t j = 0; j < GPUid.size(); ++j) {
            hipSetDevice(GPUid[j]);
            digitStream2[i][j].init();
        }
    }
}

#endif

const std::vector<int> Context::generateDigitGPUid(std::vector<std::vector<LimbRecord>>& meta, const int L,
                                                   const int dnum) {
    std::vector<int> res(dnum);
    for (size_t i = 0; i < meta.size(); ++i) {
        for (auto& j : meta[i]) {
            res[j.digit] = i;
        }
    }
    return res;
}

std::vector<std::vector<LimbRecord>> Context::generateSplitSpecialMeta(std::vector<LimbRecord>& specialMeta,
                                                                       const std::vector<int> GPUid) {
    std::vector<std::vector<LimbRecord>> res(GPUid.size());

    int init = 0;
    for (int i = 0; i < GPUid.size(); ++i) {
        hipSetDevice(GPUid[i]);
        int num = (specialMeta.size() - init) / (GPUid.size() - i);
        for (int j = init; j < init + num; ++j) {
            res[i].emplace_back(
                LimbRecord{.id = specialMeta[j].id, .type = specialMeta[j].type, .digit = specialMeta[j].digit});
            res[i].back().stream.init();
        }
        init += num;
    }
    return res;
}

std::vector<Ciphertext> bootAuxCipher;

Context::~Context() {
    CudaCheckErrorMod;
    eval_key.reset();
    rot_keys.clear();
    boot_precomps.clear();
    key_switch_aux.reset(nullptr);
    key_switch_aux2.reset(nullptr);
    bootAuxCipher.clear();
    for (auto& i : moddown_aux) {
        i.reset(nullptr);
    }

    for (size_t i = 0; i < top_limbptr.size(); ++i) {
        hipSetDevice(GPUid[i]);
#ifdef NCCL
        NCCLCHECK(ncclCommDeregister(GPUrank[i], top_limb_buffer_handle[i]));
        NCCLCHECK(ncclCommDeregister(GPUrank[i], top_limb_buffer2_handle[i]));
        NCCLCHECK(ncclMemFree(top_limb_buffer[i]));
        NCCLCHECK(ncclMemFree(top_limb_buffer2[i]));
#endif
        top_limbptr[i].free(top_limb_stream[i]);
        top_limbptr2[i].free(top_limb_stream2[i]);
    }
#ifdef NCCL
    ncclGroupStart();
    for (auto& rank : devices) {
        if (rank.second) {
            hipSetDevice(rank.first);
            NCCLCHECK(ncclCommFinalize(*rank.second));
            NCCLCHECK(ncclCommDestroy(*rank.second));
        }
    }
    ncclGroupEnd();

    devices.clear();
#endif
    Ciphertext::clearOpRecord();
    CudaCheckErrorMod;
}

std::vector<Ciphertext>& Context::getBootstrapAuxilarCiphertexts() {
    return bootAuxCipher;
}

}  // namespace FIDESlib::CKKS
