//
// Created by carlosad on 25/04/24.
//

#include <cereal/external/rapidjson/internal/itoa.h>
#include "CKKS/Ciphertext.cuh"
#include "CKKS/Context.cuh"
#include "CKKS/Plaintext.cuh"

namespace FIDESlib::CKKS {

Plaintext::Plaintext(Context& cc)
    : my_range(loc, LIFETIME),
      cc((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))), cc)),
      c0(cc) {
    CudaNvtxStop();
}

Plaintext::Plaintext(Context& cc, const RawPlainText& raw)
    : my_range(loc, LIFETIME),
      cc((CudaNvtxStart(std::string{std::source_location::current().function_name()}.substr(18 + strlen(loc))), cc)),
      c0(cc) {
    load(raw);
    CudaNvtxStop();
}

void Plaintext::load(const RawPlainText& raw) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    c0.loadConstant(raw.sub_0, raw.moduli);

    NoiseFactor = raw.Noise;
    NoiseLevel = raw.NoiseLevel;
    /*
    hipDeviceSynchronize();
    std::cout << "plaintext ";
    for (auto& j : c0.GPU) {
        hipSetDevice(j.device);

        for (auto& k : j.limb) {
            SWITCH(k, printThisLimb(1));
        }
    }
    std::cout << std::endl;
    for (auto& j : c0.GPU) {
        hipSetDevice(j.device);
        for (auto& k : j.SPECIALlimb) {
            SWITCH(k, printThisLimb(1));
        }
    }
    std::cout << std::endl;
    hipDeviceSynchronize();
    */
}

void Plaintext::store(RawPlainText& raw) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    hipDeviceSynchronize();

    raw.numRes = c0.getLevel() + 1;
    raw.sub_0.resize(raw.numRes);
    c0.store(raw.sub_0);
    raw.N = cc.N;
    c0.sync();

    raw.Noise = NoiseFactor;
    raw.NoiseLevel = NoiseLevel;
    hipDeviceSynchronize();
}

void Plaintext::moddown() {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    c0.moddown(true, true);
}

bool Plaintext::adjustPlaintextToCiphertext(const Plaintext& p, const Ciphertext& c) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    constexpr bool PRINT = false;

    if (cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (p.c0.getLevel() - p.NoiseLevel > c.getLevel() - c.NoiseLevel) {
            this->copy(p);
            if (c.NoiseLevel == 1 && NoiseLevel == 2) {
                this->c0.dropToLevel(c.getLevel() + 1);
                rescale();
            } else {
                this->c0.dropToLevel(c.getLevel());
            }
            return true;
        } else if (c.NoiseLevel == 1 && p.NoiseLevel == 2) {
            this->copy(p);
            rescale();
            return true;
        } else if (p.NoiseLevel == 1 && c.NoiseLevel == 2) {
            return false;
        } else {
            this->copy(p);
            return true;
        }
    }
    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        usint c1lvl = p.c0.getLevel();
        usint c2lvl = c.getLevel();
        usint c1depth = p.NoiseLevel;
        usint c2depth = c.NoiseLevel;
        auto sizeQl1 = c1lvl + 1;
        auto sizeQl2 = c2lvl + 1;

        if (c1lvl > c2lvl) {
            this->copy(p);
            if (c1depth == 2) {
                if (c2depth == 2) {
                    double scf1 = NoiseFactor;
                    double scf2 = c.NoiseFactor;
                    double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                    double q1 =
                        cc.param.ModReduceFactor[sizeQl1 - 1];  // cryptoParams->GetModReduceFactor(sizeQl1 - 1);
                    multScalar(scf2 / scf1 * q1 / scf, false);
                    rescale();
                    if (c1lvl > c2lvl) {
                        this->c0.dropToLevel(c2lvl);
                        //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 1);
                    }
                    NoiseFactor *= scf2 / scf1 * q1 / scf;
                    assert(NoiseFactor == c.NoiseFactor);
                    NoiseFactor = c.NoiseFactor;
                } else {
                    if (c1lvl - 1 == c2lvl) {
                        rescale();
                    } else {
                        double scf1 = NoiseFactor;
                        double scf2 =
                            cc.param
                                .ScalingFactorRealBig[c2lvl + 1];  //cryptoParams->GetScalingFactorRealBig(c2lvl - 1);
                        double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                        double q1 =
                            cc.param.ModReduceFactor[sizeQl1 - 1];  //cryptoParams->GetModReduceFactor(sizeQl1 - 1);
                        multScalar(scf2 / scf1 * q1 / scf, false);
                        rescale();
                        if (c1lvl - 2 > c2lvl) {
                            this->c0.dropToLevel(c2lvl + 1);
                            //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 2);
                        }
                        rescale();

                        NoiseFactor = c.NoiseFactor;
                    }
                }
            } else {
                if (c2depth == 2) {
                    double scf1 = NoiseFactor;
                    double scf2 = c.NoiseFactor;
                    double scf = cc.param.ScalingFactorReal[c1lvl];  // cryptoParams->GetScalingFactorReal(c1lvl);
                    multScalar(scf2 / scf1 / scf, false);
                    this->c0.dropToLevel(c2lvl);
                    //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl);
                    NoiseFactor = scf2;
                } else {
                    if constexpr (PRINT)
                        std::cout << "Adjusting plaintext with noiseDegree 1" << std::endl;
                    double scf1 = NoiseFactor;
                    double scf2 =
                        cc.param.ScalingFactorRealBig[c2lvl + 1];    //cryptoParams->GetScalingFactorRealBig(c2lvl - 1);
                    double scf = cc.param.ScalingFactorReal[c1lvl];  //cryptoParams->GetScalingFactorReal(c1lvl);
                    if constexpr (PRINT)
                        std::cout << "Scale adjustment: " << scf << std::endl;

                    multScalar(scf2 / scf1 / scf, false);
                    if (c1lvl - 1 > c2lvl) {
                        if constexpr (PRINT)
                            std::cout << "Dropping levels: " << c1lvl - c2lvl - 1 << std::endl;
                        this->c0.dropToLevel(c2lvl + 1);
                        //LevelReduceInternalInPlace(ciphertext1, c2lvl - c1lvl - 1);
                    }
                    rescale();
                    NoiseFactor = c.NoiseFactor;
                }
            }
            return true;
        } else if (c1lvl < c2lvl) {
            return false;
        } else {
            this->copy(p);
            if (c1depth < c2depth) {
                multScalar(1.0, false);
            } else if (c2depth < c1depth) {
                rescale();
            }
            return true;
        }
    }
    return false;
}
void Plaintext::copy(const Plaintext& p) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    this->c0.copy(p.c0);
    this->NoiseFactor = p.NoiseFactor;
    this->NoiseLevel = p.NoiseLevel;
}
void Plaintext::multScalar(double c, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    /*
    if (cc.rescaleTechnique == Context::FLEXIBLEAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT ||
        cc.rescaleTechnique == Context::FIXEDAUTO) {
        if (NoiseLevel == 2)
            this->rescale();
    }
    assert(this->NoiseLevel == 1);
    */
    auto elem = cc.ElemForEvalMult(c0.getLevel(), c);
    /*
    for (int i = 0; i < elem.size(); i++) {
        std::cout << elem[i] << " ";
    }
    std::cout << std::endl;
*/
    c0.multScalar(elem);

    if (rescale) {
        c0.rescale();
    }
    // Manage metadata
    NoiseLevel += 1;
    NoiseFactor *= cc.param.ScalingFactorReal.at(c0.getLevel() + rescale);
    if (rescale) {
        NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + rescale);
        NoiseLevel -= 1;
    }
}

void Plaintext::rotate_hoisted(const std::vector<int>& indexes, std::vector<Plaintext*>& results) {
    assert(indexes.size() == results.size() && "rotate_hoisted: mismatched indexes and results sizes");

    for (size_t i = 0; i < indexes.size(); ++i) {
        int index = indexes[i];
        if (index == 0) {
            results[i]->copy(*this);
        } else {

            // Copy and rotate
            results[i]->copy(*this);
            results[i]->automorph(index);
        }
    }
}

#if false
void Plaintext::multPt(const Plaintext& b, bool rescale) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (cc.rescaleTechnique == Context::FIXEDAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTO ||
        cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        if (NoiseLevel == 2)
            this->rescale();
    }

    if (cc.rescaleTechnique == Context::FIXEDAUTO || cc.rescaleTechnique == Context::FLEXIBLEAUTO ||
        cc.rescaleTechnique == Context::FLEXIBLEAUTOEXT) {
        // if (b.c0.getLevel() != this.getLevel() || b.NoiseLevel == 2 /*!hasSameScalingFactor(b)*/) {
        if (!hasSameScalingFactor(b)) {
            Plaintext b_(cc);
            if (NoiseLevel == 2)
                this->rescale();
            if (b_.NoiseLevel == 2)
                b_.rescale();
            multPt(b_, rescale);
            return;
        }
    }

    assert(NoiseLevel < 2);
    assert(b.NoiseLevel < 2);
    c0.multPt(b.c0, rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL);

    // Manage metadata
    NoiseLevel += b.NoiseLevel;
    NoiseFactor *= b.NoiseFactor;
    if (rescale && cc.rescaleTechnique == CKKS::Context::FIXEDMANUAL) {
        NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
        NoiseLevel -= 1;
    }
}

void Plaintext::addPt(const Plaintext& c) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    // assert(NoiseLevel == b.NoiseLevel);
    c0.add(c.c0);
}

#endif

void Plaintext::rescale() {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));
    assert(this->NoiseLevel >= 2);
    /*
    std::cout << "Rescale plaintext, level" << c0.getLevel() << std::endl;
    for (auto& i : c0.GPU) {
        std::cout << i.limb.size() << " ";
    }
    std::cout << std::endl;
*/
    c0.rescale();

    // Manage metadata
    NoiseFactor /= cc.param.ModReduceFactor.at(c0.getLevel() + 1);
    NoiseLevel -= 1;
}

void Plaintext::automorph(const int index) {
    CudaNvtxRange r(std::string{std::source_location::current().function_name()}.substr(23 + strlen(loc)));

    if (c0.isModUp()) {
        std::cout << "isModup plaintext automorph not implemented" << std::endl;
    }
    auto& aux = cc.getModdownAux(0);
    aux.setLevel(c0.getLevel());
    aux.automorph(index, 1, &c0);
    c0.copy(aux);
}

}  // namespace FIDESlib::CKKS